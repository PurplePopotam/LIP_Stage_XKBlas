#include "hip/hip_runtime.h"
#include <cassert>
#include <>
#include <chrono>
#include "kernels.cuh"
#include <iostream>


bool check(myFloat* v, myFloat* w, unsigned int N, double epsilon) {
	double errMax = 0.0;
	for (size_t i = 0; i < N; i++)
	{
		errMax = max(abs(v[i] - w[i]), errMax);
		if (errMax > epsilon) {
			return false;
		}
	}
	return true;
}

void initVec(myFloat* v, unsigned int N, myFloat min, myFloat max) {
	for (size_t i = 0; i < N; i++)
	{
		v[i] = rand() % int(max) + myFloat(rand()) / RAND_MAX + min;
	}
}

void display(myFloat* v, unsigned int N) {

	for (size_t i = 0; i < N; i++)
	{
		std::cout << v[i] << " ";
	}
	std::cout << "\n";
}



int main(int argc, char** argv) {

#define N atoi(argv[1])
#define debug 0

	size_t bytesMatrix = sizeof(myFloat) * N * N;
	size_t bytesVector = sizeof(myFloat) * N;
	dim3 BLOCK_SIZE(THREADS_NUMBER, THREADS_NUMBER, 1);
	dim3 GRID_SIZE((N / THREADS_NUMBER), (N / THREADS_NUMBER), 1);

	hipEvent_t startGPU, stopGPU;
	hipEventCreate(&startGPU); hipEventCreate(&stopGPU);
	float milliseconds;
	std::chrono::duration<double, std::milli> millisecondsCPU;

	Matrix A(N);
	myFloat* v,* resCPU, *resGPU;
	myFloat* d_A,* d_v,* d_res;

	hipMalloc((void**)&d_A, bytesMatrix);
	hipMalloc((void**)&d_v, bytesVector);
	hipMalloc((void**)&d_res, bytesVector);

	v = new myFloat[N];
	resCPU = new myFloat[N];
	resGPU = new myFloat[N]
	A.randMatrix(0, 1);
	initVec(v, N, 0, 1);


	hipMemcpy((void*)d_A, (void*)A->content, bytesMatrix, hipMemcpyHostToDevice);
	hipMemcpy((void*)d_v, (void*)v, bytesVector, hipMemcpyHostToDevice);
	hipMemcpy((void*)d_res, (void*)resGPU, bytesVector, hipMemcpyHostToDevice);

	//CPU dense matrix/vector product
	auto startCPU = std::chrono::high_resolution_clock::now();
	resCPU = A * v;
	auto stopCPU = std::chrono::high_resolution_clock::now();

	millisecondsCPU = stopCPU - startCPU;

	//GPU dense matrix/vector product
	hipEventRecord(startGPU);
	matrixVectorV1 << <GRID_SIZE, BLOCK_SIZE >> > (d_A, d_v, d_res, N);
	hipEventRecord(stopGPU);

	hipEventSynchronize(stopGPU);

	hipMemcpy((void*)resGPU, (void*)d_res, bytesVector, hipMemcpyDeviceToHost);

	hipEventElapsedTime(&milliseconds, startGPU, stopGPU);

	if (debug) {
		if (check(resCPU, resGPU, 0.001)) {
			std::cout << "The operation is correct.\n\n"
		}
		else {
			std::cout << "The operation is incorrect.\n\n"
		}
		A.display(N);
		display(v, N);
		display(resCPU, N);
		display(resGPU, N);
	}

	std::cout << "Iteration " << " | " << "CPU matrix/vector exec time" << " | " << "GPU matrix/vector exec time" << " | " << "GPU spmv exec time\n\n";
	std::cout << "    1     " << " |    " << millisecondsCPU.count() << "   |    " << milliseconds << "    |  \n\n";


	return 0;
}
