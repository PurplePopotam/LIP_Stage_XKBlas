﻿#include <cassert>
#include <>
#include <chrono>
#include "kernels.cuh"
#include <iostream>
#include "mkl.h"

void check(myFloat* A, myFloat* B, unsigned int N) {
	for (size_t i = 0; i < N; i++)
	{
		for (size_t j = 0; j < N; j++)
		{
			assert(A[i * N + j] == B[i * N + j]);
		}
	}
}

void display(myFloat* A, unsigned int N){
	for(size_t i = 0; i < N; i++){
		for(size_t j = 0; j < N; j++){
			std::cout << A[i * N + j] << " ";
		}
		std::cout << "\n";
	}
	std::cout << "\n";
}

int main(int argc, char** argv) {


#define N atoi(argv[1])
#define debug 0
	
	size_t bytes = sizeof(myFloat) * N * N;
	dim3 BLOCK_SIZE(THREADS_NUMBER, THREADS_NUMBER, 1);
	dim3 GRID_SIZE((N/THREADS_NUMBER) + 1, (N / THREADS_NUMBER) + 1, 1);

	std::chrono::duration<double, std::milli> millisecondsCPUhost;

	hipEvent_t startGPU;
	hipEvent_t stopGPU;
	hipEventCreate(&startGPU);
	hipEventCreate(&stopGPU);
	float milliseconds;
	
	double alpha, beta;

	Matrix* h_A, * h_B, * h_C;
	myFloat* d_A, * d_B, * d_C;
	myFloat* cpu_A, *  cpu_B, *  cpu_C;

	h_A = new Matrix(N);
	h_B = new Matrix(N);
	h_C = new Matrix(N);

	hipMalloc((void**)&d_A, bytes);
	hipMalloc((void**)&d_B, bytes);
	hipMalloc((void**)&d_C, bytes);

	cpu_A = (myFloat*)mkl_malloc(bytes, 64);
	cpu_B = (myFloat*)mkl_malloc(bytes, 64);
	cpu_C = (myFloat*)mkl_malloc(bytes, 64);
	
	if (cpu_A == NULL || cpu_B == NULL || cpu_C == NULL) {
		std::cout << "\n ERROR: Can't allocate memory for matrices. Aborting... \n\n";
		mkl_free(cpu_A);
		mkl_free(cpu_B);
		mkl_free(cpu_C);
		return 1;
	}	
	
	alpha = 1.0;
	beta = 0.0;

	std::cout << "Initializing Matrix data...\n\n";

	*h_A = Matrix::randMatrix(N);
	*h_B = Matrix::randMatrix(N);
	*h_C = Matrix::nullMatrix(N);

	for(size_t i = 0; i < N*N; i++){
		cpu_A[i] = rand()%10;
		cpu_B[i] = rand()%10;	
		cpu_C[i] = rand()%10;
	}

	hipMemcpy((void*)d_A, (void*)h_A->content, bytes, hipMemcpyHostToDevice);
	hipMemcpy((void*)d_B, (void*)h_B->content, bytes, hipMemcpyHostToDevice);
	hipMemcpy((void*)d_C, (void*)h_C->content, bytes, hipMemcpyHostToDevice);

	//CPU Matrix Multiplication
	auto startCPUhost = std::chrono::high_resolution_clock::now();
	cblas_dgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans, N, N, N, alpha, cpu_A, N, cpu_B, N, beta, cpu_C, N);
	auto stopCPUhost = std::chrono::high_resolution_clock::now();

	millisecondsCPUhost = stopCPUhost - startCPUhost;
	
	//GPU Matrix Multiplication
	hipEventRecord(startGPU);
	matrixMulV3<<<GRID_SIZE, BLOCK_SIZE>>> (d_A, d_B, d_C, N);
	hipEventRecord(stopGPU);
	

	hipEventSynchronize(stopGPU);

	hipMemcpy(h_C->content, (void*)d_C, bytes, hipMemcpyDeviceToHost);

	hipEventElapsedTime(&milliseconds, startGPU, stopGPU);

	if (debug) {
		std::cout << "CPU result : " << std::endl << std::endl;
		display(cpu_C, N);
		std::cout << "GPU result : " << std::endl << std::endl;
		h_C->display();
	}
	
	//check(h_C->content,cpu_C, N);
	
	mkl_free(cpu_A); mkl_free(cpu_B); mkl_free(cpu_C);
	hipFree((void*)d_A); hipFree((void*)d_B); hipFree((void*)d_C);
	d_A = nullptr; d_B = nullptr; d_C = nullptr;

	std::cout << std::endl << "Matrix multiplication of " << N << " elements took " << millisecondsCPUhost.count() << " ms to complete on the CPU. " << std::endl << std::endl;
	std::cout << std::endl << "Matrix multiplication of " << N << " elements took " << milliseconds << " ms to complete on the GPU. " << std::endl << std::endl;
	//Test push multiple remotes
	return 0;
} 
