﻿#include <cassert>
#include <>
#include <chrono>
#include "kernels.cuh"
#include <iostream>


int main(int argc, char** argv) {

#define N atoi(argv[1])

#define debug 1
	
	size_t bytes = sizeof(myFloat) * N * N;
	dim3 BLOCK_SIZE(N, N, 1);

	std::chrono::duration<double, std::milli> millisecondsCPUhost;

	hipEvent_t startGPU;
	hipEvent_t stopGPU;
	hipEventCreate(&startGPU);
	hipEventCreate(&stopGPU);
	float milliseconds;

	Matrix* h_A, * h_B, * h_C, * hostRes_C;
	myFloat* d_A, * d_B, * d_C;

	h_A = new Matrix(N);
	h_B = new Matrix(N);
	h_C = new Matrix(N);
	hostRes_C = new Matrix(N);

	hipMalloc((void**)&d_A, bytes);
	hipMalloc((void**)&d_B, bytes);
	hipMalloc((void**)&d_C, bytes);

	*h_A = Matrix::idMatrix(N);
	*h_B = Matrix::idMatrix(N);
	*h_C = Matrix::nullMatrix(N);
	*hostRes_C = Matrix::nullMatrix(N);

	hipMemcpy((void*)d_A, (void*)h_A->content, bytes, hipMemcpyHostToDevice);
	hipMemcpy((void*)d_B, (void*)h_B->content, bytes, hipMemcpyHostToDevice);
	hipMemcpy((void*)d_C, (void*)h_C->content, bytes, hipMemcpyHostToDevice);

	
	auto startCPUhost = std::chrono::high_resolution_clock::now();
	*hostRes_C = *h_A + *h_B;
	auto stopCPUhost = std::chrono::high_resolution_clock::now();

	millisecondsCPUhost = stopCPUhost - startCPUhost;

	hipEventRecord(startGPU);
	matrixAddV1 <<<1, BLOCK_SIZE >>> (d_A, d_B, d_C, N);
	hipEventRecord(stopGPU);

	hipEventSynchronize(stopGPU);

	hipMemcpy(h_C->content, (void*)d_C, bytes, hipMemcpyDeviceToHost);
	

	hipEventElapsedTime(&milliseconds, startGPU, stopGPU);

	if (debug) {
		std::cout << "CPU result : " << std::endl << std::endl;
		hostRes_C->display();
		std::cout << "GPU result : " << std::endl << std::endl;
		h_C->display();
	}
	

	std::cout << std::endl << "Matrix addition of " << N << " elements took " << millisecondsCPUhost.count() << " ms to complete on the CPU. " << std::endl << std::endl;
	std::cout << std::endl << "Matrix addition of " << N << " elements took " << milliseconds << " ms to complete on the GPU. " << std::endl << std::endl;

	return 0;
}