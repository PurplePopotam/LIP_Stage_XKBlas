﻿#include <cassert>
#include <>
#include <chrono>
#include "kernels.cuh"
#include <iostream>

void check(myFloat* A, myFloat* B, unsigned int N) {
	for (size_t i = 0; i < N; i++)
	{
		for (size_t j = 0; j < N; j++)
		{
			assert(A[i * N + j] == B[i * N + j]);
		}
	}
}

void display(myFloat* A, unsigned int N){
	for(size_t i = 0; i < N; i++){
		for(size_t j = 0; j < N; j++){
			std::cout << A[i * N + j] << " ";
		}
		std::cout << "\n";
	}
	std::cout << "\n";
}

int main(int argc, char** argv) {


#define N atoi(argv[1])
#define debug 0
	
	size_t bytes = sizeof(myFloat) * N * N;
	dim3 BLOCK_SIZE(THREADS_NUMBER, THREADS_NUMBER, 1);
	dim3 GRID_SIZE((N/THREADS_NUMBER) + 1, (N / THREADS_NUMBER) + 1, 1);

	hipEvent_t startGPU, startGPUtiled;
	hipEvent_t stopGPU, stopGPUtiled;
	hipEventCreate(&startGPUtiled); hipEventCreate(&startGPU);
	hipEventCreate(&stopGPUtiled); hipEventCreate(&stopGPU);
	float millisecondsTiled, milliseconds;

	Matrix* h_A, * h_B, * h_C, * h_C_tiled;
	myFloat* d_A, * d_B, * d_C, * d_C_tiled;

	h_A = new Matrix(N);
	h_B = new Matrix(N);
	h_C = new Matrix(N);
	h_C_tiled = new Matrix(N);

	hipMalloc((void**)&d_A, bytes);
	hipMalloc((void**)&d_B, bytes);
	hipMalloc((void**)&d_C, bytes);
	hipMalloc((void**)&d_C_tiled, bytes);

	std::cout << "Initializing Matrix data...\n\n";

	*h_A = Matrix::randMatrix(N);
	*h_B = Matrix::randMatrix(N);
	*h_C = Matrix::nullMatrix(N);

	std::cout << "Done initialazing. \n\n";

	hipMemcpy((void*)d_A, (void*)h_A->content, bytes, hipMemcpyHostToDevice);
	hipMemcpy((void*)d_B, (void*)h_B->content, bytes, hipMemcpyHostToDevice);
	hipMemcpy((void*)d_C, (void*)h_C->content, bytes, hipMemcpyHostToDevice);
	hipMemcpy((void*)d_C_tiled, (void*)h_C->content, bytes, hipMemcpyHostToDevice);

	//GPU tiled Matrix Multiplication
	hipEventRecord(startGPUtiled);
	matrixMulV2<<<GRID_SIZE, BLOCK_SIZE>>> (d_A, d_B, d_C_tiled, N);
	hipEventRecord(stopGPUtiled);
	
	hipEventSynchronize(stopGPUtiled);

	//GPU regular Matrix Multiplication with small optimizations
	hipEventRecord(startGPU);
	matrixMulV4i<<<GRID_SIZE, BLOCK_SIZE>>> (d_A, d_B, d_C, N);
	hipEventRecord(stopGPU);

	hipMemcpy(h_C_tiled->content, (void*)d_C_tiled, bytes, hipMemcpyDeviceToHost);

	hipEventElapsedTime(&millisecondsTiled, startGPUtiled, stopGPUtiled);

	if (debug) {
		std::cout << "GPU result : " << std::endl << std::endl;
		h_C->display();
		std::cout << "GPU tiled result : " << std::endl << std::endl;
		h_C_tiled->display();
	}
	
	check(h_C->content,h_C_tiled->content, N);
	
	hipFree((void*)d_A); hipFree((void*)d_B); hipFree((void*)d_C);
	d_A = nullptr; d_B = nullptr; d_C = nullptr;

	std::cout << std::endl << "Matrix multiplication of " << N << " elements took " << millisecondsTiled << " ms to complete on the GPU. " << std::endl << std::endl;

	return 0;
} 
