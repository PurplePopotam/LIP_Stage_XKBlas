﻿#include <cassert>
#include <>
#include <chrono>
#include "kernels.cuh"
#include <iostream>


void check(myFloat* A, myFloat* B, unsigned int N) {
	for (size_t i = 0; i < N; i++)
	{
		for (size_t j = 0; j < N; j++)
		{
			assert(A[i * N + j] == B[i * N + j]);
		}
	}
}

int main(int argc, char** argv) {


#define N atoi(argv[1])
#define debug 0
	
	size_t bytes = sizeof(myFloat) * N * N;
	dim3 BLOCK_SIZE(THREADS_NUMBER, THREADS_NUMBER, 1);
	dim3 GRID_SIZE((N/THREADS_NUMBER) + 1, (N / THREADS_NUMBER) + 1, 1);

	std::chrono::duration<double, std::milli> millisecondsCPUhost;

	hipEvent_t startGPU;
	hipEvent_t stopGPU;
	hipEventCreate(&startGPU);
	hipEventCreate(&stopGPU);
	float milliseconds;

	Matrix* h_A, * h_B, * h_C, * hostRes_C;
	myFloat* d_A, * d_B, * d_C;

	h_A = new Matrix(N);
	h_B = new Matrix(N);
	h_C = new Matrix(N);
	hostRes_C = new Matrix(N);

	hipMalloc((void**)&d_A, bytes);
	hipMalloc((void**)&d_B, bytes);
	hipMalloc((void**)&d_C, bytes);

	*h_A = Matrix::randMatrix(N);
	*h_B = Matrix::randMatrix(N);
	*h_C = Matrix::nullMatrix(N);
	*hostRes_C = Matrix::nullMatrix(N);

	hipMemcpy((void*)d_A, (void*)h_A->content, bytes, hipMemcpyHostToDevice);
	hipMemcpy((void*)d_B, (void*)h_B->content, bytes, hipMemcpyHostToDevice);
	hipMemcpy((void*)d_C, (void*)h_C->content, bytes, hipMemcpyHostToDevice);

	//CPU Matrix Multiplication
	auto startCPUhost = std::chrono::high_resolution_clock::now();
	*hostRes_C = *h_A * *h_B;	
	auto stopCPUhost = std::chrono::high_resolution_clock::now();

	millisecondsCPUhost = stopCPUhost - startCPUhost;
	
	//GPU Matrix Multiplication
	hipEventRecord(startGPU);
	matrixMulV1 <<<GRID_SIZE, BLOCK_SIZE >>> (d_A, d_B, d_C, N);
	hipEventRecord(stopGPU);
	

	hipEventSynchronize(stopGPU);

	hipMemcpy(h_C->content, (void*)d_C, bytes, hipMemcpyDeviceToHost);

	hipEventElapsedTime(&milliseconds, startGPU, stopGPU);

	if (debug) {
		std::cout << "CPU result : " << std::endl << std::endl;
		hostRes_C->display();
		std::cout << "GPU result : " << std::endl << std::endl;
		h_C->display();
	}
	
	check(hostRes_C->content, h_C->content, N);

	std::cout << std::endl << "Matrix multiplication of " << N << " elements took " << millisecondsCPUhost.count() << " ms to complete on the CPU. " << std::endl << std::endl;
	std::cout << std::endl << "Matrix multiplication of " << N << " elements took " << milliseconds << " ms to complete on the GPU. " << std::endl << std::endl;

	return 0;
} 