﻿#include <cassert>
#include <>
#include <chrono>
#include "kernels.cuh"
#include <iostream>

void check(myFloat* A, myFloat* B, unsigned int N) {
	for (size_t i = 0; i < N; i++)
	{
		for (size_t j = 0; j < N; j++)
		{
			assert(A[i * N + j] == B[i * N + j]);
		}
	}
}

void display(myFloat* A, unsigned int N){
	for(size_t i = 0; i < N; i++){
		for(size_t j = 0; j < N; j++){
			std::cout << A[i * N + j] << " ";
		}
		std::cout << "\n";
	}
	std::cout << "\n";
}

int main(int argc, char** argv) {


#define N atoi(argv[1])
#define debug 0
	
	size_t bytes = sizeof(myFloat) * N * N;
	dim3 BLOCK_SIZE(THREADS_NUMBER, THREADS_NUMBER, 1);
	dim3 GRID_SIZE((N/THREADS_NUMBER) + 1, (N / THREADS_NUMBER) + 1, 1);

	hipEvent_t startGPU, startGPUtiled;
	hipEvent_t stopGPU, stopGPUtiled;
	hipEventCreate(&startGPUtiled); hipEventCreate(&startGPU);
	hipEventCreate(&stopGPUtiled); hipEventCreate(&stopGPU);
	float millisecondsTiled, milliseconds;
	std::chrono::duration<double, std::milli> millisecondsCPUinit;

	Matrix* h_A, * h_B, * h_C, * h_C_tiled;
	myFloat* d_A, * d_B, * d_C, * d_C_tiled;

	h_A = new Matrix(N);
	h_B = new Matrix(N);
	h_C = new Matrix(N);
	h_C_tiled = new Matrix(N);

	hipMalloc((void**)&d_A, bytes);
	hipMalloc((void**)&d_B, bytes);
	hipMalloc((void**)&d_C, bytes);
	hipMalloc((void**)&d_C_tiled, bytes);

	std::cout << "Initializing Matrix data...\n\n";
	auto startCPU = std::chrono::high_resolution_clock::now();
	h_A->randMatrix(0, 10);
	h_B->randMatrix(0, 10);
	h_C->nullMatrix();
	h_C_tiled->nullMatrix();
	auto stopCPU = std::chrono::high_resolution_clock::now();
	std::cout << "Done initialazing. \n\n";

	millisecondsCPUinit = stopCPU - startCPU;

	std::cout << "Init took " << millisecondsCPUinit.count() << " ms.\n\n";
	hipMemcpy((void*)d_A, (void*)h_A->content, bytes, hipMemcpyHostToDevice);
	hipMemcpy((void*)d_B, (void*)h_B->content, bytes, hipMemcpyHostToDevice);
	hipMemcpy((void*)d_C, (void*)h_C->content, bytes, hipMemcpyHostToDevice);
	hipMemcpy((void*)d_C_tiled, (void*)h_C_tiled->content, bytes, hipMemcpyHostToDevice);
	
	//GPU tiled Matrix Multiplication
	hipEventRecord(startGPUtiled);
	matrixMulV2<<<GRID_SIZE, BLOCK_SIZE>>> (d_A, d_B, d_C_tiled, N);
	hipEventRecord(stopGPUtiled);
	
	hipEventSynchronize(stopGPUtiled);

	hipMemcpy((void*)h_C_tiled->content, (void*)d_C_tiled, bytes, hipMemcpyDeviceToHost);

	//GPU regular Matrix Multiplication with small optimizations
	hipEventRecord(startGPU);
	matrixMulV4f<<<GRID_SIZE, BLOCK_SIZE>>> (d_A, d_B, d_C, N);
	hipEventRecord(stopGPU);
	
	hipEventSynchronize(stopGPU);

	hipMemcpy((void*)h_C->content, (void*)d_C, bytes, hipMemcpyDeviceToHost);

	hipEventElapsedTime(&millisecondsTiled, startGPUtiled, stopGPUtiled);
	hipEventElapsedTime(&milliseconds, startGPU, stopGPU);

	if (debug) {
		std::cout << "GPU result : \n\n";
		h_C->display();
		std::cout << "GPU tiled result : \n\n";
		h_C_tiled->display();
	}
	
	//check(h_C->content,h_C_tiled->content, N);
	
	//Freeing the memory
	
	hipFree((void*)d_A); hipFree((void*)d_B); hipFree((void*)d_C); hipFree((void*)d_C_tiled);
	free(h_A); free(h_B); free(h_C); free(h_C_tiled);
	d_A = nullptr; d_B = nullptr; d_C = nullptr; d_C_tiled = nullptr;
	h_A = nullptr; h_B = nullptr; h_C = nullptr; h_C_tiled = nullptr;
	
	std::cout << std::endl << "Tiled matrix multiplication of " << N << " elements took " << millisecondsTiled << " ms to complete on the GPU.\n\n";
	std::cout << std::endl << "Regular matrix multiplication of " << N << " elements took " << milliseconds << " ms to complete on the GPU.\n\n";
	return 0;
} 
