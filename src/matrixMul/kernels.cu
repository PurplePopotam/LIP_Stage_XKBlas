#include "hip/hip_runtime.h"
﻿#include "kernels.cuh"
#include <iostream>


__global__ void matrixAddV1(myFloat* A, myFloat* B, myFloat* C, unsigned int N) {

	unsigned int tidX = threadIdx.x + blockDim.x * blockIdx.x;
	unsigned int tidY = threadIdx.y + blockDim.y * blockIdx.y;

	if (tidX < N && tidY < N) {
		C[tidY * N + tidX] = A[tidY * N + tidX] + B[tidY * N + tidX];
	}
	
}

__global__ void matrixMulV3(myFloat* A, myFloat* B, myFloat* C, unsigned int N) {
	unsigned int tidX = threadIdx.x + blockDim.x * blockIdx.x;
	unsigned int tidY = threadIdx.y + blockDim.y * blockIdx.y;
	unsigned int strideX = blockDim.x * gridDim.x;
	unsigned int strideY = blockDim.y * gridDim.y;

	myFloat tmp = 0;

	while(tidX < N && tidY < N) {
		for (size_t k = 0; k < N; k+=4)
		{
			float4
			tmp += A[tidY * N + k] * B[k * N + tidX];
		}
		C[tidY * N + tidX] = tmp;
		tidX += strideX;
		tidY += strideY;
	}
}

__global__ void matrixMulV4f(float* A, float* B, float* C, unsigned int N) {
	unsigned int tidX = threadIdx.x + blockDim.x * blockIdx.x;
	unsigned int tidY = threadIdx.y + blockDim.y * blockIdx.y;
	unsigned int strideX = blockDim.x * gridDim.x;
	unsigned int strideY = blockDim.y * gridDim.y;

	myFloat tmp = 0;

	while (tidX < N && tidY < N) {
		for (size_t k = 0; k < N; k += 4)
		{
			float4 a_tmp = reinterpret_cast<float4*>(&a[row * N + i])[0];
			tmp += a_tmp.x * B[k * N + tidX];
			tmp += a_tmp.y * B[(k + 1) * N + tidX];
			tmp += a_tmp.z * B[(k + 1) * N + tidX];
			tmp += a_tmp.w * B[(k + 1) * N + tidX];
		}
		C[tidY * N + tidX] = tmp;
		tidX += strideX;
		tidY += strideY;
	}
}