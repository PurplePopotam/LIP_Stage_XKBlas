#include "hip/hip_runtime.h"
﻿#include "kernels.cuh"
#include <iostream>


__global__ void matrixAddV1(myFloat* A, myFloat* B, myFloat* C, unsigned int N) {

	unsigned int tidX = threadIdx.x + blockDim.x * blockIdx.x;
	unsigned int tidY = threadIdx.y + blockDim.y * blockIdx.y;
	unsigned int strideX = blockDim.x * gridDim.x;
	unsigned int strideY = blockDim.y * gridDim.y;

	if (tidX < N && tidY < N) {
		C[tidY * N + tidX] = A[tidY * N + tidX] + B[tidY * N + tidX];
	}
	
}

__global__ void matrixMulV1(myFloat* A, myFloat* B, myFloat* C, unsigned int N) {
	//global IDs
	unsigned int tidX = threadIdx.x + blockDim.x * blockIdx.x;
	unsigned int tidY = threadIdx.y + blockDim.y * blockIdx.y;

	if (tidX < N && tidY < N) {
		for (size_t k = 0; k < N; k++)
		{
			C[tidY * N + tidX] +=  A[tidY * N + k] * B[k * N + tidX];
		}
	}
}

__global__ void matrixMulV2(myFloat* A, myFloat* B, myFloat* C, unsigned int N) {
	__shared__ myFloat cache_A[THREADS_NUMBER * THREADS_NUMBER];
	__shared__ myFloat cache_B[THREADS_NUMBER * THREADS_NUMBER];

	unsigned int tidX = threadIdx.x + THREADS_NUMBER * blockIdx.x;
	unsigned int tidY = threadIdx.y + THREADS_NUMBER * blockIdx.y;

	myFloat tmp = 0;

	for (size_t i = 0; i < N/THREADS_NUMBER; i++)
	{
		cache_A[threadIdx.y * THREADS_NUMBER + threadIdx.x] = A[tidY * N + i * THREADS_NUMBER + threadIdx.x];
		cache_B[threadIdx.y * THREADS_NUMBER + threadIdx.x] = B[(i * THREADS_NUMBER + threadIdx.y) * N + tidX];
		__syncthreads();

		for (size_t k = 0; k < THREADS_NUMBER; k++)
		{
			tmp += cache_A[threadIdx.y * THREADS_NUMBER + k] * cache_B[k * THREADS_NUMBER + threadIdx.x];
			__syncthreads();
		}
		
	}
	C[tidY * N + tidX] = tmp;
}
__global__ void MatrixMulKernel(float* Md, float* Nd, float* Pd, int Width) {
	__shared__ float Mds[THREADS_NUMBER][THREADS_NUMBER];
	__shared__ float Nds[THREADS_NUMBER][THREADS_NUMBER];
	int bx = blockIdx.x; int by = blockIdx.y;
	int tx = threadIdx.x; int ty = threadIdx.y;
	// Identify the row and column of the Pd element to work on
	int Row = by * THREADS_NUMBER + ty;
	int Col = bx * THREADS_NUMBER + tx;
	float Pvalue = 0;
	// Loop over the Md and Nd tiles required to compute the Pd element
	for (int m = 0; m < Width / THREADS_NUMBER; ++m) {
		// Collaborative loading of Md and Nd tiles into shared memory
		Mds[ty][tx] = Md[Row * Width + (m * THREADS_NUMBER + tx)];
		Nds[ty][tx] = Nd[Col + (m * THREADS_NUMBER + ty) * Width];

	 __syncthreads();
		for (int k = 0; k < THREADS_NUMBER; ++k) {
			Pvalue += Mds[ty][k] * Nds[k][tx];
			__syncthreads();
		}
	}
	Pd[Row * Width + Col] = Pvalue;
}

__global__ void matrixMulV3(myFloat* A, myFloat* B, myFloat* C, unsigned int N) {
	unsigned int tidX = threadIdx.x + blockDim.x * blockIdx.x;
	unsigned int tidY = threadIdx.y + blockDim.y * blockIdx.y;
	unsigned int strideX = blockDim.x * gridDim.x;
	unsigned int strideY = blockDim.y * gridDim.y;

	myFloat tmp = 0;

	while(tidX < N && tidY < N) {
		for (size_t k = 0; k < N; k++)
		{
			tmp += A[tidY * N + k] * B[k * N + tidX];
		}
		C[tidY * N + tidX] = tmp;
		tidX += strideX;
		tidY += strideY;
	}

	
}