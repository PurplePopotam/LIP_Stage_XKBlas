﻿#include <stdio.h>
#include <cassert>
#include <iostream>
#include <>
#include <chrono>
#include "kernels.cuh"


#define N 4000000
#define THREADS_PER_BLOCK 512
float cpuDotProduct(float* a, float* b, int n) {
    float res = 0.0f;

    for (int i = 0; i < n; i++) {
        res += a[i] * b[i];
    }

    return res;
}


int main() {

    //CudaEvents are used to measure the execution time on the GPU
    hipEvent_t startGPU;
    hipEvent_t stopGPU;
    hipEventCreate(&startGPU);
    hipEventCreate(&stopGPU);
    float milliseconds;



    float* h_a, * h_b, * h_c;
    float* d_a, * d_b, * d_c;

    h_a = new float[N];
    h_b = new float[N];
    h_c = new float;

    size_t bytes = sizeof(float) * N;

    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, sizeof(float));

    for (size_t i = 0; i < N; i++) {
        h_a[i] = rand() % 10;
        h_b[i] = rand() % 10;
    }
    *h_c = 0;

    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_c, h_c, sizeof(float), hipMemcpyHostToDevice);

    int NUM_THREADS = 512;
    int NUM_BLOCKS = N / THREADS_PER_BLOCK;

    //Doing the dot product on the device
    hipEventRecord(startGPU);
    dotProductV2 <<<NUM_BLOCKS, NUM_THREADS>>> (d_a, d_b, d_c, N);
    hipEventRecord(stopGPU);

    hipMemcpy(h_c, d_c, sizeof(float), hipMemcpyDeviceToHost);

    hipEventSynchronize(stopGPU);

    //Doing the dot product on the host
    auto startCPU = std::chrono::high_resolution_clock::now();
    float res = cpuDotProduct(h_a, h_b, N);
    auto stopCPU = std::chrono::high_resolution_clock::now();

    std::chrono::duration<double, std::milli> millisecondsCPU = stopCPU - startCPU;

    hipEventElapsedTime(&milliseconds, startGPU, stopGPU);
    

    std::cout << "Dot product results on CPU : " << res << std::endl;
    std::cout << "Dot product results on GPU : " << *h_c << std::endl;
    std::cout << std::endl;
    std::cout << "GPU execution time : " << milliseconds << " ms . " << std::endl;
    //std::cout << "GPU bandwidth : " << N * 6 / milliseconds / 1e6 << " GB/s ." << std::endl;    //Not the accurate numbre of flops 
    std::cout << std::endl;
    std::cout << "CPU execution time : " << millisecondsCPU.count() << " ms ." << std::endl;    
    std::cout << "CPU bandwidth : " << N * 3 / millisecondsCPU.count() / 1e6 << " GB/s ." << std::endl; //3 W/R operations in the cpuDotProduct function

    std::cout << std::endl;
    return 0;
}