﻿#include <stdio.h>
#include <cassert>
#include <iostream>
#include <>
#include <chrono>
#include "kernels.cuh"


#define ITER 10

//Prise de temps CPU des kernels 
//Prise de temps séparemment des transfert de données
//Boucle pour mesurer sur plusieurs lancements
//

myFloat cpuDotProduct(myFloat* a, myFloat* b, int n) {
    myFloat res = 0.0f;

    for (int i = 0; i < n; i++) {
        res += a[i] * b[i];
    }

    return res;
}


int main(int argc, char** argv) {

    #define N atoi(argv[1])
    //CudaEvents are used to measure the execution time on the GPU
    bool debug = false;

    hipEvent_t startGPU;
    hipEvent_t stopGPU;
    hipEventCreate(&startGPU);
    hipEventCreate(&stopGPU);
    float milliseconds;

    std::chrono::duration<double, std::milli> millisecondsCPUhost;
    std::chrono::duration<double, std::milli> millisecondsCPUdevice;
    std::chrono::duration<double, std::milli> millisecondsDeviceHostCopy;

    
    size_t bytes = sizeof(myFloat) * N;

    std::cout << "DotProduct of " << N/1000000 << "M elements, using " << THREADS_PER_BLOCK << " threads per block. " << std::endl;
    std::cout << std::endl;
    std::cout << "Iteration " << " | " << "CPU host exec time" << " | " << "GPU device exec time" << " | " << "CPU device exec time" << " | " << "device -> host copy duration" << std::endl;

    for (size_t i = 0; i < ITER; i++) {

        myFloat* h_a, * h_b, * h_c;
        myFloat* d_a, * d_b, * d_c;
        myFloat resCPU;

        h_a = new myFloat[N];
        h_b = new myFloat[N];
        h_c = new myFloat;

        hipMalloc((void**)&d_a, bytes);
        hipMalloc((void**)&d_b, bytes);
        hipMalloc((void**)&d_c, sizeof(myFloat));

        for (size_t j = 0; j < N; j++) {
            h_a[j] = rand() % 10;
            h_b[j] = rand() % 10;
        }
        *h_c = 0;

        hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
        hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);
        hipMemcpy(d_c, h_c, sizeof(myFloat), hipMemcpyHostToDevice);

        int NUM_BLOCKS = N / THREADS_PER_BLOCK;

        //Doing the dot product on the host

        auto startCPUhost = std::chrono::high_resolution_clock::now();
        resCPU = cpuDotProduct(h_a, h_b, N);
        auto stopCPUhost = std::chrono::high_resolution_clock::now();
        
        millisecondsCPUhost = stopCPUhost - startCPUhost;

        //Doing the dot product on the device

        auto startCPUdevice = std::chrono::high_resolution_clock::now();
        hipEventRecord(startGPU);

        dotProductV3 << <NUM_BLOCKS, THREADS_PER_BLOCK >> > (d_a, d_b, d_c, N);

        hipEventRecord(stopGPU);
        hipEventSynchronize(stopGPU);
        auto stopCPUdevice = std::chrono::high_resolution_clock::now();

        millisecondsCPUdevice = stopCPUdevice - startCPUdevice;

        auto startDeviceHostCopy = std::chrono::high_resolution_clock::now();
        hipMemcpy(h_c, d_c, sizeof(myFloat), hipMemcpyDeviceToHost);
        auto stopDeviceHostCopy = std::chrono::high_resolution_clock::now();

        millisecondsDeviceHostCopy = stopDeviceHostCopy - startDeviceHostCopy;

        hipEventElapsedTime(&milliseconds, startGPU, stopGPU);

        if (debug) {
            std::cout << "Dot product results on GPU : " << *h_c << std::endl;
            std::cout << "Dot product results on CPU : " << resCPU << std::endl;
            std::cout << std::endl;
        }

        //Results display

        
        std::cout << std::endl;
        std::cout << "     " << i << "    " << " |     " << millisecondsCPUhost.count() << " ms " << "    |       " << milliseconds << " ms " << "    |      " << millisecondsCPUdevice.count() << " ms " << "      |      " << millisecondsDeviceHostCopy.count() << " ms" << std::endl;

        hipFree(&d_a); hipFree(&d_b); hipFree(&d_c);
        delete(h_a); delete(h_b); delete(h_c);
    }

    return 0;
}